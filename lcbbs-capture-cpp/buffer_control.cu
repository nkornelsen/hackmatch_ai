#include "shared.hpp"
#include <hip/hip_runtime.h>
#include <cuda_d3d11_interop.h>
#include <cudaD3D11.h>
#include <d3d11.h>
#include <stdio.h>

void cuda_init() {
    return;
}

CudaState new_cuda_state(IDXGIAdapter* adapter) {

    CudaState s{0};
    cudaD3D11GetDevice(&s.dev, adapter);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, s.dev);
    printf("CUDA Device: %s\n", prop.name);
    return s;
}

void set_graphics_resource(CudaState &s, ID3D11Resource* resource) {
    auto e = cudaGraphicsD3D11RegisterResource(&s.resource, resource, hipGraphicsRegisterFlagsNone);
    printf("CUDA Buffer Access: %s\n", hipGetErrorName(e));
}